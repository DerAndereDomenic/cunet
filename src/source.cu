#include "hip/hip_runtime.h"
#include "../lib/include/CUNET.h"
#include "TestModel.h"


__global__ void test(cunet::TestModel mlp)
{
    printf("Start\n");
    float input[] = { 1,2,3,4,5,6,7,8,9,10 };
    cunet::Tensor<float> inp;
    inp.setData<10>(input);
    cunet::Tensor<float>& output = mlp(inp);

    for(uint32_t i = 0; i < output.size(); ++i)
    {
        printf("%f\n", output[i]);
    }
    printf("End\n");
}

int main()
{
    cunet::TestModel mlp;

    test << <1, 1 >> > (mlp);
    cudaSafeCall(hipDeviceSynchronize());

    return 0;
}